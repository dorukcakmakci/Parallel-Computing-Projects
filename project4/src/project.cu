#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <ctype.h>

#include "hip/hip_runtime.h"

// kernel function for matrix vector product computation
__global__ void computeMatrixVectorProduct(int *d_row_ptr, int *d_col_ind, double *d_vector, double * d_result_vector, double* d_values, int num_of_threads, int matrix_dim, int num_of_ele) {
    
    // first each thread computes which portion of the data to work on which is represented as [start_index, end_index)
    int start_index, end_index;

    start_index = 0;
    end_index = 0;

    int quotient = matrix_dim / num_of_threads;
    int remainder = matrix_dim % num_of_threads;
    int number_count = quotient;

    if(threadIdx.x < remainder) {
        number_count = quotient + 1;
    }

    for (int i = 0; i < threadIdx.x; i++) {
        if (i < remainder) {
            start_index += quotient + 1;
        }
        else {
            start_index += quotient;
        }
    }

    end_index = start_index + number_count;

    if(end_index > matrix_dim) {
        end_index = matrix_dim;
    }

    int cur_col_ind;
    int next_col_ind;

    for (int row = start_index; row < end_index; row++) {

        d_result_vector[row] = 0;

        // last row
        if(row == matrix_dim - 1) {

            cur_col_ind = d_row_ptr[row];

            while(cur_col_ind < num_of_ele && cur_col_ind >= 0) {
                d_result_vector[row] += d_values[d_col_ind[cur_col_ind]] * d_vector[d_col_ind[cur_col_ind]];
                cur_col_ind++;
            }
        }

        // empty row
        else if(d_row_ptr[row] == d_row_ptr[row + 1]){
            d_result_vector[row] = 0;
        }

        // if all the following rows are empty
        else if( d_row_ptr[row] >= num_of_ele ) {
            d_result_vector[row] = 0;
        }

        // and other case
        else{

            cur_col_ind = d_row_ptr[row];
            next_col_ind = d_row_ptr[row + 1];

            while( cur_col_ind  < next_col_ind) {
                d_result_vector[row] += d_values[d_col_ind[cur_col_ind]] * d_vector[d_col_ind[cur_col_ind]];
                cur_col_ind++;
            }

        }
    }

    
}

int main(int argc, char ** argv) {

    int thread_count = atoi(argv[1]);
    int iter_count = atoi(argv[2]);
    int flag = atoi(argv[3]);
    char *filename = argv[4];

    // read sparse matrix file and initialize row_ptr, col_ind and values arrays
    // the following code assumes input matrix is a square matrix
    FILE *matrix = fopen(filename, "r");
    int num_cols, num_rows, num_ele;
    fscanf(matrix, "%d %d %d\n", &num_rows, &num_cols, &num_ele);
    int row_ptr[num_rows];

    // initialize index counter and row_ptr
    int counter[num_rows];
    for (int i = 0; i < num_rows; i++) {
        counter[i] = 0;
        row_ptr[i] = 0;
    }

    // first read the file to find the number of non-zero elements in each row
    int cur_row, cur_col;
    double cur_val;

    // the maximum number of non-zero values is given in the mtx file but the actual numbe rof elements may be less than that
    // so hold an actual count of values
    int actual_count = 0;

    while (EOF != fscanf(matrix, "%d %d %le\n", &cur_row, &cur_col, &cur_val)) {
        cur_row -= 1;
        cur_col -= 1;
        row_ptr[cur_row]++;
        actual_count++;
    } 

    int col_ind[actual_count];
    double values[actual_count];

    // convert row_ptr to an increasing array by adding an element and the element before that.
    int  temp = row_ptr[0];
    int temp_2;
    row_ptr[0] = 0;

    for(int j = 1; j <= num_rows - 1; j++) {
        temp_2 = row_ptr[j];
        row_ptr[j] = row_ptr[j-1] + temp;
        temp = temp_2;
    }

    fclose(matrix);

    // read file again to initialize values
    matrix = fopen(filename, "r");

    // discard first line 
    fscanf(matrix, "%d %d %d\n", &num_rows, &num_cols, &num_ele);

    while (EOF != fscanf(matrix, "%d %d %le\n", &cur_row, &cur_col, &cur_val)) {
        cur_row -= 1;
        cur_col -= 1;
        if( cur_row < num_rows - 1 ) {
            temp = row_ptr[cur_row] + counter[cur_row];
            if(temp < row_ptr[cur_row + 1]) {
                col_ind[temp] = cur_col;
                values[temp] = cur_val;
            }
            counter[cur_row]++;
        }
        else {
                temp = row_ptr[cur_row] + counter[cur_row];
                if(temp < actual_count) {
                    col_ind[temp] = cur_col;
                    values[temp] = cur_val;
                }
                counter[cur_row]++;
        }
    }

    double *x = (double *) malloc(num_rows * sizeof(double));
    for( int i = 0; i < num_rows; i++) {
        x[i] = 1;
    }

    // print sparse matrix and vector before computation
    if(flag == 1){
        printf("Before Computation:\n");

        printf("Vector X: \n");
        for(int i = 0; i < num_rows; i++) {
            printf("%le ", x[i]);
        }
        printf("\n");

        

        int i;
        printf("Initial Matrix: \n");
        printf("row_ptr array: \n");
        for (i = 0; i <= num_rows - 1; i++) {
            printf("%d ", row_ptr[i]);
        }
        printf("\n");

        printf("col_ind array: \n");
        for (i = 0; i <= actual_count - 1; i++) {
            printf("%d ", col_ind[i]);
        }
        printf("\n");

        printf("values array: \n");
        for (i = 0; i <= actual_count - 1; i++) {
            printf("%le ", values[i]);
        }
        printf("\n");
    }

    // computation starts here

    // initialize device related structures 
    double *d_values, *d_vector, *d_result_vector;
    int *d_row_ptr, *d_col_ind;

    hipMalloc(&d_values, actual_count * sizeof(double));
    hipMalloc(&d_vector, num_rows * sizeof(double));
    hipMalloc(&d_result_vector, num_rows * sizeof(double));
    hipMalloc(&d_row_ptr, num_rows * sizeof(int));
    hipMalloc(&d_col_ind, actual_count * sizeof(int));

    hipMemcpy(d_values, values, actual_count * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_col_ind, col_ind, actual_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, row_ptr, num_rows * sizeof(int), hipMemcpyHostToDevice);

    int iter;
    for(iter = 0; iter < iter_count; iter++) {
        hipMemcpy(d_vector, x, num_rows * sizeof(double), hipMemcpyHostToDevice);

        // call kernel code for sparse matrix and vector multiplication
        computeMatrixVectorProduct<<<1, thread_count>>>(d_row_ptr, d_col_ind, d_vector, d_result_vector, d_values, thread_count, num_rows, actual_count);

        hipMemcpy(x, d_result_vector, num_rows * sizeof(double), hipMemcpyDeviceToHost);

        // free device matrices

        // // for testing purposes
        // // printf("1: %s\n", cudaGetErrorString(cudaGetLastError()));
        // printf("Iter: %d, Vector X: \n", iter);
        // for(int i = 0; i < num_rows; i++) {
        //     printf("%le ", x[i]);
        // }
        // printf("\n");
    }

    hipFree(d_values);
    hipFree(d_vector);
    hipFree(d_row_ptr);
    hipFree(d_col_ind);
    hipFree(d_result_vector);

    // print resulting vector from computation
    if(flag == 1 || flag == 2){

        printf("After Computation:\n");

        printf("Vector X: \n");
        for(int i = 0; i < num_rows; i++) {
            printf("%le ", x[i]);
        }
        printf("\n");
    }

    return 0;

}